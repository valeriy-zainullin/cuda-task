#include "hip/hip_runtime.h"
#include <Filter.cuh>

#include <CommonKernels.cuh>

#include <cstdio>

// __device__ static int dev_max(int a, int b) {
//     if (a > b) {
//        return a;
//    }
//    return b;
// }

// COPYPASTE from common kernels.
//   Can't call __device__ functions from other
//   object files. Why? It took a lot of time
//   from me. And it's not even relevant..

__device__
void DevDoIntScan1(int *array, int array_size) {
    int num_threads = blockDim.x;
    int thread_index = threadIdx.x;

    for (int step = 1; (1 << step) <= array_size; ++step) {
        int item_array_pos = thread_index * (1 << step);
        for (int item = thread_index; item < array_size / (1 << step); item += num_threads) {
            int prev_step_shift = (1 << step) / 2;
            array[item_array_pos] += array[item_array_pos + prev_step_shift];
            item_array_pos += num_threads * (1 << step);
        }
        __syncthreads();
    }
}

__device__
void DevDoIntScan2(int *array, int array_size) {
    int num_threads = blockDim.x;
    int thread_index = threadIdx.x;

    for (int step_size = array_size; step_size >= 2; step_size /= 2) {
        int item_array_pos = thread_index * step_size;
        int next_item_array_pos = item_array_pos + step_size;
        // Картинка помогает понять ситуацию (прямо моя реализация):
        //   https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf,
        //   страница 8.
        // На самом верхнем уровне один элемент, у него уже правильное значение.
        //   Хотим из дерева получить префиксные суммы.
        //   На каждом уровне хранится только правый ребенок, левый
        //   перезаписан значением родителя. Так что чинить нужно только
        //   Правых детей. Правого ребенка, отрезок которого примыкакает к
        //   концу массива, чинить не нужно. Если не примыкает, добавляем
        //   соседа нашего уровня. Тогда получится префиксная сумма, т.к.
        //   теперь отрезок учтенных элементов примыкает к концу массива.
        for (int item = thread_index; item < array_size / step_size - 1; item += num_threads) {
            int prev_step_shift = step_size / 2;
//            int prev_left = item_array_pos;
            int prev_right = item_array_pos + prev_step_shift;

            array[prev_right] += array[next_item_array_pos];

            item_array_pos += num_threads * step_size;
            next_item_array_pos += num_threads * step_size;
        }
        __syncthreads();
    }
}

// ---- END OF COPYPASTE. ----

__global__ void Filter(
    int num_items,
    float* array,
    OperationFilterType op_type,
    float* value,
    float* result,
    float* aux_array1_float,
    float* aux_array2_float
) {
    int* aux_array1 = (int*) aux_array1_float;
    int* aux_array2 = (int*) aux_array2_float;

    uint3 thread_index_3d = threadIdx;
    int thread_index_in_block = thread_index_3d.x;

    uint3 block_dim_3d = blockDim;
    int num_threads_per_block = block_dim_3d.x;

    uint3 grid_dim_3d = gridDim;
    int num_blocks  = grid_dim_3d.x;

    uint3 block_index_3d = blockIdx;
    int block_index = block_index_3d.x; // В 0-индексации.

    int thread_index = block_index * num_threads_per_block + thread_index_in_block;
    int num_threads  = num_blocks * num_threads_per_block;

    if (op_type == LT) {
        for (int item_index = thread_index; item_index < num_items; item_index += num_threads) {
            if (array[item_index] < value[item_index]) {
                aux_array1[item_index] = 1;
                aux_array2[item_index] = 1;
            } else {
                aux_array1[item_index] = 0;
                aux_array2[item_index] = 0;
            }
        }
    } else /* if (op_type == GT) */ {
        for (int item_index = thread_index; item_index < num_items; item_index += num_threads) {
            if (array[item_index] > value[item_index]) {
                aux_array1[item_index] = 1;
                aux_array2[item_index] = 1;
            } else {
                aux_array1[item_index] = 0;
                aux_array2[item_index] = 0;
            }
        }
    }

//    __syncthreads();
//    for (int item_index = thread_index; item_index < num_items; item_index += num_threads) {
//        printf("item_index = %d, aux_array2[item_index] = %d, value = %.0f.\n", item_index, aux_array2[item_index], value[item_index]);
//    }

    // Дожидаемся, пока все потоки завершат отсеивать элементы.
    __syncthreads();

    // Считаем сумму на префиксах, так поймем, на какое место встает элемент.
    //   Разделим потоки по позициям исходного массива, они будут проставлять
    //   этот элемент на нужную позицию. Эта позиция принадлежит только этому
    //   элементу, потому все ок.
    // В нашем случае, сумма на суффиксах, т.е. знаем позицию с конца для всех,
    //   кроме последнего. Его проставляет первый поток.
    DevDoIntScan1(aux_array1, num_items);
    __syncthreads();
    DevDoIntScan2(aux_array1, num_items);

//    __syncthreads();
//    for (int item_index = thread_index; item_index < num_items; item_index += num_threads) {
//        printf("item_index = %d, aux_array1[item_index] = %d, value = %.0f.\n", item_index, aux_array1[item_index], value[item_index]);
//    }

    __syncthreads();

    for (int item_index = thread_index; item_index < num_items; item_index += num_threads) {
        if (aux_array2[item_index]) {
            result[aux_array1[0] - aux_array1[item_index]] = array[item_index];
        }
    }
}

