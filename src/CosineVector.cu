#include "hip/hip_runtime.h"
#include <CosineVector.cuh>

#include <ScalarMulRunner.cuh>

#include <cmath>

float CosineVector(int num_items, float* vector1, float* vector2, int block_size) {
    // (v1, v2) = |v_1| |v_2| cos(v_1 ^ v_2)
    float scalar_prod = ScalarMulSumPlusReduction(num_items, vector1, vector2, block_size);

    float vector1_len2 = ScalarMulSumPlusReduction(num_items, vector1, vector1, block_size);
    float vector1_len = std::sqrt(vector1_len2);

    float vector2_len2 = ScalarMulSumPlusReduction(num_items, vector2, vector2, block_size);
    float vector2_len = std::sqrt(vector2_len2);

    float angle = scalar_prod / (vector1_len * vector2_len);

    return angle;
}

