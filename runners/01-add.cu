#include "KernelAdd.cuh"

#include "OnExit.h"

#include <algorithm>
#include <iostream>
#include <iomanip>

template <typename T>
static T* alloc_copy_from_host(T* src, size_t src_len) {
    static_assert(std::is_trivially_copyable_v<T>);

    hipError_t status = hipSuccess;

    T* dst = nullptr;
    status = hipMalloc(&dst, src_len * sizeof(T));
    if (status != hipSuccess) {
        std::cerr << "Failed to allocate memory.\n";
        return nullptr;
    }

    status = hipMemcpy(dst, src, src_len * sizeof(T), hipMemcpyHostToDevice);
    if (status != hipSuccess) {
        std::cerr << "Failed to copy from host to device.\n";
        hipFree(dst);
        return nullptr;
    }

    return dst;
}

template <typename T>
static T* alloc_memset(int byte, size_t num_bytes) {
    hipError_t status = hipSuccess;

    T* dst = nullptr;
    status = hipMalloc(&dst, num_bytes);
    if (status != hipSuccess) {
        std::cerr << "Failed to allocate memory.\n";
        return nullptr;
    }

    status = hipMemset(dst, byte, num_bytes);
    if (status != hipSuccess) {
        std::cerr << "Failed to memset device memory.\n";
        hipFree(dst);
        return nullptr;
    }

    return reinterpret_cast<T*>(dst);
}


int main() {
    int num_blocks = 4;
    int num_threads_per_block = 4;

    float items1[] = {1, 2, 3, 4, 5, 6};
    float items2[] = {3, 4, 5, 6, 7, 8};
    constexpr size_t num_items1 = sizeof(items1) / sizeof(*items1);
    constexpr size_t num_items2 = sizeof(items2) / sizeof(*items2);
    static_assert(num_items1 == num_items2);

    constexpr size_t num_items = num_items1;

    float* dev_items1 = alloc_copy_from_host(items1, num_items1);
    if (dev_items1 == nullptr) {
        std::cerr << "Failed to copy items1 to device.\n";
        return 1;
    }
    ON_EXIT({
        hipFree(dev_items1);
    });

    float* dev_items2 = alloc_copy_from_host(items2, num_items2);
    if (dev_items2 == nullptr) {
        std::cerr << "Failed to copy items2 to device.\n";
        return 2;
    }
    ON_EXIT({
        hipFree(dev_items2);
    });

    float result[num_items] = {};
    float* dev_result = alloc_memset<float>(0, sizeof(float) * num_items);
    if (dev_result == nullptr) {
        std::cerr << "Failed to create result array on the device.\n";
        return 3;
    }
    ON_EXIT({
        hipFree(dev_result);
    });

    // https://en.wikipedia.org/wiki/Thread_block_(CUDA_programming)
    //   (Thread) block is a set of threads, threads from the same
    //   block are executed on the same stream processor (stream
    //   processor is a set of cores, gpu consists of stream
    //   processors).
    KernelAdd<<<num_blocks, num_threads_per_block>>>(static_cast<int>(num_items), dev_items1, dev_items2, dev_result);

    hipError_t status = hipMemcpy(result, dev_result, sizeof(result), hipMemcpyDeviceToHost);
    if (status != hipSuccess) {
        std::cerr << "Failed to copy result array to the host.\n";
        return 4;
    }

    for (size_t i = 0; i < num_items; ++i) {
        std::cout << std::fixed << std::setw(1);
        std::cout << items1[i] << " + " << items2[i] << " = " << result[i] << '\n';
    }

    return 0;
}
