#include "hip/hip_runtime.h"
#include <Filter.cuh>

#include <CommonKernels.cuh>
#include <OnExit.h>
#include <ScalarMul.cuh>

#include <algorithm>
#include <iostream>
#include <iomanip>
#include <type_traits>

template <typename T>
static T* alloc_copy_from_host(T* src, size_t src_len) {
    static_assert(std::is_trivially_copyable_v<T>);

    hipError_t status = hipSuccess;

    T* dst = nullptr;
    status = hipMalloc(&dst, src_len * sizeof(T));
    if (status != hipSuccess) {
        std::cerr << "Failed to allocate memory.\n";
        return nullptr;
    }

    status = hipMemcpy(dst, src, src_len * sizeof(T), hipMemcpyHostToDevice);
    if (status != hipSuccess) {
        std::cerr << "Failed to copy from host to device.\n";
        hipFree(dst);
        return nullptr;
    }

    return dst;
}

template <typename T>
static T* alloc_memset(int byte, size_t num_bytes) {
    hipError_t status = hipSuccess;

    T* dst = nullptr;
    status = hipMalloc(&dst, num_bytes);
    if (status != hipSuccess) {
        std::cerr << "Failed to allocate memory.\n";
        return nullptr;
    }

    status = hipMemset(dst, byte, num_bytes);
    if (status != hipSuccess) {
        std::cerr << "Failed to memset device memory.\n";
        hipFree(dst);
        return nullptr;
    }

    return reinterpret_cast<T*>(dst);
}


int main() {
    int num_blocks = 1;
    int num_threads_per_block = 4;

    float items1[] = {1, 2, 3, 4, 5, 6, 0, 1};
    float items2[] = {3, 4, 2, 3, 1, 8, 0, 1};
    constexpr size_t num_items1 = sizeof(items1) / sizeof(*items1);
    constexpr size_t num_items2 = sizeof(items2) / sizeof(*items2);
    static_assert(num_items1 == num_items2);

    constexpr size_t num_items = num_items1;

    float* dev_items1 = alloc_copy_from_host(items1, num_items1);
    if (dev_items1 == nullptr) {
        std::cerr << "Failed to copy items1 to device.\n";
        return 1;
    }
    ON_EXIT({
        hipFree(dev_items1);
    });

    float* dev_items2 = alloc_copy_from_host(items2, num_items2);
    if (dev_items2 == nullptr) {
        std::cerr << "Failed to copy items2 to device.\n";
        return 2;
    }
    ON_EXIT({
        hipFree(dev_items2);
    });

    float result[num_items] = {};
    float* dev_result = alloc_memset<float>(0, sizeof(float) * num_items);
    if (dev_result == nullptr) {
        std::cerr << "Failed to create result array on the device.\n";
        return 3;
    }
    ON_EXIT({
        hipFree(dev_result);
    });

    float* dev_aux_array1 = alloc_memset<float>(0, sizeof(float) * num_items);
    if (dev_aux_array1 == nullptr) {
        std::cerr << "Failed to create aux_array1 on the device.\n";
        return 3;
    }
    ON_EXIT({
        hipFree(dev_aux_array1);
    });

    float* dev_aux_array2 = alloc_memset<float>(0, sizeof(float) * num_items);
    if (dev_aux_array2 == nullptr) {
        std::cerr << "Failed to create aux_array2 on the device.\n";
        return 3;
    }
    ON_EXIT({
        hipFree(dev_aux_array2);
    });

    std::cout << "items1 = [";
    for (size_t i = 0; i < num_items; ++i) {
        std::cout << std::fixed << std::setw(1);
        if (i != 0) {
            std::cout << ", ";
        }
        std::cout << items1[i];
    }
    std::cout << "]\n";

    std::cout << "items2 = [";
    for (size_t i = 0; i < num_items; ++i) {
        std::cout << std::fixed << std::setw(1);
        if (i != 0) {
            std::cout << ", ";
        }
        std::cout << items2[i];
    }
    std::cout << "]\n";

    // https://en.wikipedia.org/wiki/Thread_block_(CUDA_programming)
    //   (Thread) block is a set of threads, threads from the same
    //   block are executed on the same stream processor (stream
    //   processor is a set of cores, gpu consists of stream
    //   processors).
    // If we pass ints, it creates uint3 gridDim, uint3 blockDim
    //   with x equal to our values and y, z equal to 1.
    // If we pass uint3 as gridDim and blockDim,
    //   it'll create gridDim.x * gridDim.y * gridDim.z blocks,
    //   each block will have blockDim.x * blickDim.y * blockDim.z
    //   threads.
    Filter<<<num_blocks, num_threads_per_block>>>(static_cast<int>(num_items), dev_items1, LT, dev_items2, dev_result, dev_aux_array1, dev_aux_array2);

    hipError_t status = hipMemcpy(result, dev_result, sizeof(result), hipMemcpyDeviceToHost);
    if (status != hipSuccess) {
        std::cerr << "Failed to copy result array to the host.\n";
        return 4;
    }

    std::cout << "result_lt = [";
    for (size_t i = 0; i < num_items; ++i) {
        std::cout << std::fixed << std::setw(1);
        if (i != 0) {
            std::cout << ", ";
        }
        std::cout << result[i];
    }
    std::cout << "]\n";

    Filter<<<num_blocks, num_threads_per_block>>>(static_cast<int>(num_items), dev_items1, GT, dev_items2, dev_result, dev_aux_array1, dev_aux_array2);

    status = hipMemcpy(result, dev_result, sizeof(result), hipMemcpyDeviceToHost);
    if (status != hipSuccess) {
        std::cerr << "Failed to copy result array to the host.\n";
        return 4;
    }

    std::cout << "result_gt = [";
    for (size_t i = 0; i < num_items; ++i) {
        std::cout << std::fixed << std::setw(1);
        if (i != 0) {
            std::cout << ", ";
        }
        std::cout << result[i];
    }
    std::cout << "]\n";

    return 0;
}
